
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <string>

using namespace std;

// Code to get hypotenuse of a triangle
// The parts of the array are [a, b, c]
__global__
void hypotenuse(float *sides) {
    sides[2] = sqrt((sides[0]*sides[0])+(sides[1]*sides[1]));
}

int main() {
    // Setting up an array to hold the values
    float *sides;
    string str;

    // Allocating memory on the GPU
    hipMallocManaged(&sides, 3*sizeof(float));

    // Reading in the values
    ifstream indata;
    indata.open("../Unit01/Exercise05/Inputs/input.txt");
    for (int i = 0; i < 2; i++) {
    indata >> str >> sides[i];
    }
    indata.close();

    // Getting the hypotenuse
    hypotenuse<<<1,1>>>(sides);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Writing out the result
    ofstream outdata;
    outdata.open("../Unit01/Exercise05/Outputs/output.txt");
    outdata << "c: " << sides[2] << endl;
    outdata.close();

    // Free memory
    hipFree(sides);

    return 0;


}