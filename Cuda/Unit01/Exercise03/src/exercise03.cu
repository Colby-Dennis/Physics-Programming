
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

// Trying the use of a pointer to see what happens
__global__
void square(float *n) {
    n[0] = n[0]*n[0];
}

int main() {
    // Setting the variable
    float *num;

    // Allocating memory on GPU
    hipMallocManaged(&num, sizeof(float));

    // Reading in the value
    std::ifstream indata;
    indata.open("../Unit01/Exercise03/Inputs/input.txt");
    indata >> num[0];
    indata.close();

    // Squaring the value
    square<<<1,1>>>(num);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Writing out the result
    std::cout << num[0] << std::endl;

    // Freeing memory
    hipFree(num);

    return 0;
}