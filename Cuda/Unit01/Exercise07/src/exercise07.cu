
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <fstream>

using namespace std;

// Program to square input
__global__
void square(float *n) {
    n[0] = n[0] * n[0];
}

int main (int argc, char** argv) {
    // c++ way to convert strings.
    float *n;
    hipMallocManaged(&n, sizeof(float));
    n[0] = std::stof(argv[1]);

    // Squaring the value
    square<<<1,1>>>(n);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Writing out the result
    ofstream outdata;
    outdata.open("../Unit01/Exercise07/Outputs/output.txt");
    outdata << n[0] << endl;
    outdata.close();

    // Free memory
    hipFree(n);

    return 0;
}