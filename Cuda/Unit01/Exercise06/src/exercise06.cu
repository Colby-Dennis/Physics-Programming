
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

// Program to make the square of the input (store to second array)
__global__
void square(int n, float *in, float *out) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride) {
        out[i] = in[i] * in[i];
    }
}

int main() {
    // Setting the variables
    float *x, *y;
    int n = 25;

    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));

    // Initialize the x array
    for (int i = 0; i < n; i++) {
        x[i] = float(i+1);
    }

    // Getting the squares using n threads
    square<<<1,n>>>(n, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    

    // Writing out the result in a tab delimited text file and csv
    ofstream outdataTab, outdataCsv;
    outdataTab.open("../Unit01/Exercise06/Outputs/output.txt");
    outdataCsv.open("../Unit01/Exercise06/Outputs/output.csv");
    outdataTab << "Number \t Square" << endl;
    outdataCsv << "Number,Square" << endl;
    for (int i = 0; i < n; i++) {
        outdataTab << x[i] << "\t" << y[i] << endl;
        outdataCsv << x[i] << "," << y[i] << endl;
    }
    outdataTab.close();
    outdataCsv.close();

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}