
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

// Program to make the square of the input (store to second array)
__global__
void square(int n, float *in, float *out) {
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride) {
        out[i] = in[i] * in[i];
    }
}

int main() {
    // Setting the variables
    float *x, *y;
    int n = 25;

    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));

    // Initialize the x array
    for (int i = 0; i < n; i++) {
        x[i] = float(i+1);
    }

    // Getting the squares using n threads
    square<<<1,n>>>(n, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    

    // Writing out the result
    ofstream outdata;
    outdata.open("../Unit01/Exercise06/Outputs/output.txt");
    for (int i = 0; i < n; i++) {
        outdata << x[i] << " " << y[i] << endl;
    }
    outdata.close();

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}