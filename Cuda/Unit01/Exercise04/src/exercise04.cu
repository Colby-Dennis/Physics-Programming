
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

// Don't want to keep writing std::
using namespace std;

// Function to square a number using the GPU
__global__
void square(float *n) {
    n[0] = n[0]*n[0];
}

int main() {
    // Setting the variable to get and manipulate
    float *num;

    // Allocating memory on the GPU
    hipMallocManaged(&num, sizeof(float));

    // Reading in the value
    ifstream indata;
    indata.open("../Unit01/Exercise04/Inputs/input.txt");
    indata >> num[0];
    indata.close();

    // Squaring the value
    square<<<1,1>>>(num);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Writing out the result
    ofstream outdata;
    outdata.open("../Unit01/Exercise04/Outputs/output.txt");
    outdata << num[0];
    outdata.close();

    // Freeint memory
    hipFree(num);

    return 0;
}